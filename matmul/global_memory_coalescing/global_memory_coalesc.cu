#include <hip/hip_runtime.h>
#include <iostream>


#include <hip/hip_fp16.h>
#include <mma.h>


#define BLOCKSIZE 32
#define CEIL_DIV(x, y) (((x) + (y) - 1) / (y))

//SGEMM performs C=αAB+βC at single (=32b) precision
__global__ void sgemm_naive(int M,int N,int K,float alpha,const float *A,const float *B, float beta,float *C){

    const int x=blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const int y=blockIdx.y *BLOCKSIZE + (threadIdx.x % BLOCKSIZE);
    if (x<M && y<N) {
       
        float tmp=0.f;
        for(int i=0;i<K ;++i){
            tmp+=A[x*K+i] *B[i*N+y];
        }
        C[x*N+y]=alpha*tmp +beta *C[x*N+y];    }


}



void test_sgemmen_naive(){
    const int M=4096;
    const int N=4096;
    const int K=4096;
    const float alpha=2.0;
    const float beta=3.0;

    float *A,*B,*C;
    float *d_A,*d_B,*d_C;

    A=(float*)malloc(M*K*sizeof(float));
    B=(float*)malloc(K*N*sizeof(float));
    C=(float*)malloc(M*N*sizeof(float));

    hipMalloc(&d_A,M*K*sizeof(float));
    hipMalloc(&d_B,K*N*sizeof(float));
    hipMalloc(&d_C,M*N*sizeof(float));

    for(int i=0;i<M*K;++i){
        A[i]=1.0;
    }
    for(int i=0;i<K*N;++i){
        B[i]=1.0;
    }
    for(int i=0;i<M*N;++i){
        C[i]=0.0;
    }

    hipMemcpy(d_A,A,M*K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,K*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_C,C,M*N*sizeof(float),hipMemcpyHostToDevice);

    dim3 gridDims(CEIL_DIV(M,32), CEIL_DIV(N,32));
    dim3 blockDims(32*32);

    //create timing events

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sgemm_naive<<<gridDims,blockDims>>>(M,N,K,alpha,d_A,d_B,beta,d_C);
    hipEventRecord(stop);

    //get timing 
    hipEventSynchronize(stop);
    float milliseconds=0;
    hipEventElapsedTime(&milliseconds,start,stop);

    hipMemcpy(C,d_C,M*N*sizeof(float),hipMemcpyDeviceToHost);

    printf("Time taken for sgemm_naive of %dx%d * %dx%d is %f ms\n",M,K,K,N,milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}


int main(){
    test_sgemmen_naive();
    return 0;
}