#include <hip/hip_runtime.h>
#include <iostream>


const char *ptxCode = R"(
.version 8.5
.target sm_52
.address_size 64

	// .globl	_Z3addPi

.visible .entry _Z3addPi(
	.param .u64 _Z3addPi_param_0
)
{
	.reg .b32 	%r<7>;
	.reg .b64 	%rd<5>;


	ld.param.u64 	%rd1, [_Z3addPi_param_0];
	cvta.to.global.u64 	%rd2, %rd1;
	mov.u32 	%r1, %ctaid.x;
	mov.u32 	%r2, %ntid.x;
	mov.u32 	%r3, %tid.x;
	mad.lo.s32 	%r4, %r1, %r2, %r3;
	mul.wide.s32 	%rd3, %r4, 4;
	add.s64 	%rd4, %rd2, %rd3;
	ld.global.u32 	%r5, [%rd4];
	shl.b32 	%r6, %r5, 1;
	st.global.u32 	[%rd4], %r6;
	ret;

}
)";

int main() {
    // Initialize the CUDA Driver API
    hipError_t res = hipInit(0);
    if (res != hipSuccess) {
        std::cerr << "Failed to initialize CUDA Driver API" << std::endl;
        return -1;
    }

    const int size = 5; // Array size
    int h_data[size] = {1, 2, 3, 4, 5}; // Initialize input array
    int *d_data;

    // Allocate device memory
    hipError_t err = hipMalloc((void**)&d_data, size * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Copy input data from host to device
    err = hipMemcpy(d_data, h_data, size * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy (Host to Device) failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Load PTX code
    hipModule_t module;
    hipFunction_t kernel;
    hipError_t result = hipModuleLoadData(&module, ptxCode);
    if (result != hipSuccess) {
        const char *errorStr;
        hipDrvGetErrorString(result, &errorStr);
        std::cerr << "Error loading PTX: " << errorStr << std::endl;
        return -1;
    }
    if (hipModuleGetFunction(&kernel, module, "_Z3addPi") != hipSuccess) {
        std::cerr << "Failed to get kernel function" << std::endl;
        return -1;
    }

    // Set kernel parameters
    void *args[] = { &d_data };
    dim3 grid(1);
    dim3 block(size); // One thread per element

    // Launch kernel
    hipError_t launchResult = hipModuleLaunchKernel(kernel, grid.x, grid.y, grid.z, block.x, block.y, block.z, 0, 0, args, 0);
    if (launchResult != hipSuccess) {
        std::cerr << "Kernel launch failed" << std::endl;
        return -1;
    }

    // Synchronize and check for errors
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Copy result back to host
    err = hipMemcpy(h_data, d_data, size * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy (Device to Host) failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Print result
    std::cout << "Data: ";
    for (int i = 0; i < size; ++i) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipFree(d_data);
    hipModuleUnload(module);
    return 0;
}